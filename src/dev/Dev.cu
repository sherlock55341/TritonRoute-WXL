#include "Dev.hpp"
#include <iostream>
#ifndef __NVCC__
#include <hip/hip_runtime.h>
#include <>
#include <math_functions.h>
#endif

namespace dev {
void set_device(int dev_id) {
    int gpu_count = -1;
    hipGetDeviceCount(&gpu_count);
    if (dev_id < 0 || dev_id >= gpu_count) {
        std::cout << "[CUDA ERROR] select device " << dev_id
                  << ", but there are only " << gpu_count << " devices"
                  << std::endl;
        exit(0);
    }
    hipSetDevice(dev_id);
    hipFree(0);
}
} // namespace dev